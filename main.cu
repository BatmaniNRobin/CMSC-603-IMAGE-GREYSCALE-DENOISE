#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <float.h>
#include <math.h>
#include <iostream>
#include <algorithm>
#include <tuple>
// #include <opencv4/opencv.hpp>
#include <cv.h>
#include <opencv2/core/core.hpp>
#include <opencv2/core/utility.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>

using namespace cv;
using namespace std;

#define THREADS_DIM 32
#define WINDOW_SIZE (3)


__global__ void greyscale(uchar4* d_rgb, uchar* d_grey, int matrixHeight, int matrixWidth, int numPixels)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if(col < matrixWidth && row < matrixHeight)
    {
        int rgb_ab = row * matrixWidth + col;
	    uchar4 rgb_image = d_rgb[rgb_ab];
        double gray_val = (float(rgb_image.x))*0.299f + (float(rgb_image.y))*0.587f + (float(rgb_image.z))*0.114f;
        d_grey[rgb_ab] = (unsigned char)gray_val;
    }
}

__global__ void denoise(uchar *d_grey, uchar *d_output, int matrixHeight, int matrixWidth, int numPixels)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    unsigned char array[9];

    if(col < matrixWidth && row < matrixHeight)
    {
        for(int x = 0; x < WINDOW_SIZE; x++)
        {
            for(int y = 0; y < WINDOW_SIZE; y++)
            {
                array[x*WINDOW_SIZE+y] = d_grey[(row+x-1)*matrixWidth+(col+y-1)];
            }
        }

        // TODO insertion sort
        
        int rgb_ab = row * matrixWidth + col;
        
        // int i, key, j;

        // __syncthreads();
        // // insertion sort
        // for(i = 1; i < 9; i++)
        // {
        //     key = array[i];
        //     j = i - 1;

        //     while(j >=0 && array[j] > key)
        //     {
        //         array[j+1] = array[j];
        //         j = j - 1;
        //     }
        //     array[j+1] = key;
        // }

        // // write value to d_output
        // d_output[rgb_ab] = (unsigned char) array[4];

        // bubblesort works ...
        for (int i = 0; i < 9; i++) {
            for (int j = i + 1; j < 9; j++) {
                if (array[i] > array[j]) { 
                    //Swap the variables.
                    unsigned char temp = array[i];
                    array[i] = array[j];
                    array[j] = temp;
                }
            }
        }
        d_output[rgb_ab] = (unsigned char) array[4];
    }
}


int main(int argc, char *argv[])
{
    if(argc != 1)
    {
        cout << "Usage: ./main" << endl;
        exit(0);
    }

    Mat img_RGB;
    Mat img_Grey;

    // load image into matrix obj in BGR
    Mat image = imread("image.jpg", CV_LOAD_IMAGE_COLOR);

    // cvtColor(image, BGR2RGB);

    // Check for failure
    if (image.empty()) 
    {
        cout << "Could not open or find the image" << endl;
        exit(1);
    }

    // convert color from openCV standard BGR to RGB
    cvtColor(image, img_RGB, CV_BGR2RGBA);

    // imshow("Image", image); // lol how do i know if my images are greyscale

    // // allocate memory for an imaeg to be saved that is the greyscale version that it should get saved to
    // single channel 8bit color ie grey and with uchar instead of uchar4
    img_Grey.create(image.rows, image.cols, CV_8UC1);

    // struct timespec start, end;
    float milliseconds;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // // allocate matrices on host and device
    uchar4 *d_rgb, *h_rgb;
    uchar *d_grey;

    h_rgb = (uchar4*)img_RGB.ptr<uchar>(0);

    int matrixWidth = image.cols;
    int matrixHeight = image.rows;

    int numPixels = matrixWidth * matrixHeight;

    hipMalloc(&d_rgb, numPixels * sizeof(uchar4));
    hipMalloc(&d_grey, numPixels * sizeof(uchar));

    // // transfer from host to device
    hipMemcpy(d_rgb, h_rgb, numPixels * sizeof(uchar4), hipMemcpyHostToDevice);

    hipError_t code = hipGetLastError();
    if (code != hipSuccess) 
	{
        const char * errorMessage = hipGetErrorString(code);
        printf("cuda error HtoD %s", errorMessage);
    }

    // TODO check gridDim
    // call kernel to compute matrix multiplication 
    int gridDim_x = (image.cols / THREADS_DIM) + 1;
    int gridDim_y = (image.rows / THREADS_DIM) + 1;

    dim3 blockSize (THREADS_DIM, THREADS_DIM);
    dim3 gridSize (gridDim_x, gridDim_y);

    // clock_gettime(CLOCK_MONOTONIC_RAW, &start);
    hipEventRecord(start);

    greyscale <<< gridSize, blockSize >>> (d_rgb, d_grey, matrixHeight, matrixWidth, numPixels);

    // clock_gettime(CLOCK_MONOTONIC_RAW, &end);

    // // transfer back from device to host
    hipMemcpy(img_Grey.ptr<uchar>(0), d_grey, numPixels * sizeof(uchar), hipMemcpyDeviceToHost);

    code = hipGetLastError();
    if (code != hipSuccess) 
	{
        const char * errorMessage = hipGetErrorString(code);
        printf("\ncuda error DtoH %s\n\n", errorMessage);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    imwrite("grey.jpg", img_Grey);

    printf("GPU Time for Grey: %f ms\n", milliseconds);

    //------------------------------------------------ DENOISE ---------------------------------------------------

    Mat denoised_image;

    float ms;
    hipEvent_t begin, finish;
    hipEventCreate(&begin);
    hipEventCreate(&finish);

    denoised_image.create(image.rows, image.cols, CV_8UC1);

    uchar *d_output;

    hipMalloc(&d_output, numPixels * sizeof(uchar));
    
    hipEventRecord(begin);

    denoise <<< gridSize, blockSize >>> (d_grey, d_output, matrixHeight, matrixWidth, numPixels);

    hipMemcpy(denoised_image.ptr<uchar>(0), d_output, numPixels * sizeof(uchar), hipMemcpyDeviceToHost);

    hipEventRecord(finish);
    hipEventSynchronize(finish);
    hipEventElapsedTime(&ms, begin, finish);

    imwrite("denoise.jpg", denoised_image);

    printf("GPU Time for Denoising: %f ms\n", ms);

    // // free memory
    hipFree(d_rgb);
    hipFree(d_grey);
    hipFree(d_output);
}