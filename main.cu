#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <float.h>
#include <math.h>
#include <iostream>
#include <algorithm>
#include <tuple>
// #include <opencv4/opencv.hpp>
#include <cv.h>
#include <opencv2/core/core.hpp>
#include <opencv2/core/utility.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>

using namespace cv;
using namespace std;

#define THREADS_DIM 32


__global__ void greyscale(uchar4* d_rgb, uchar* d_grey, int matrixHeight, int matrixWidth, int numPixels)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if(col < matrixWidth && row < matrixHeight)
    {
        int rgb_ab = row * matrixWidth + col;
	    uchar4 rgb_image = d_rgb[rgb_ab];
        double gray_val = (float(rgb_image.x))*0.299f + (float(rgb_image.y))*0.587f + (float(rgb_image.z))*0.114f;
        d_grey[rgb_ab] = (unsigned char)gray_val;
    }
}

__global__ void denoise(uchar *d_grey, uchar *d_output, int matrixHeight, int matrixWidth, int numPixels)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    uchar upleft, up, upright, left, center, right, downleft, down, downright;
    int i, key, j;

    __shared__ int array[9]; // prolly not 16 should be 9 but ~thread_size oooo~
    __shared__ int thread[256];

    if(col < matrixWidth && row < matrixHeight)
    {
        // up = thread[row - 1];
        // down = thread[col - 1];
    }


    if(col < matrixWidth && row < matrixHeight)
    {
        int rgb_ab = row * matrixWidth + col;

        __syncthreads();
        // insertion sort
        for(i = 1; i < 9; i++)
        {
            key = array[i];
            j = i - 1;

            while(j >=0 && array[j] > key)
            {
                array[j+1] = array[j];
                j = j - 1;
            }
            array[j+1] = key;
        }

    //     // write value to d_output
        d_output[rgb_ab] = (unsigned char) array[4];
        // d_output[rgb_ab] = d_grey[rgb_ab];
    }
}


int main(int argc, char *argv[])
{
    if(argc != 1)
    {
        cout << "Usage: ./main" << endl;
        exit(0);
    }

    Mat img_RGB;
    Mat img_Grey;

    // load image into matrix obj in BGR
    Mat image = imread("image.jpg", CV_LOAD_IMAGE_COLOR);

    // cvtColor(image, BGR2RGB);

    // Check for failure
    if (image.empty()) 
    {
        cout << "Could not open or find the image" << endl;
        exit(1);
    }

    // convert color from openCV standard BGR to RGB
    cvtColor(image, img_RGB, CV_BGR2RGBA);

    // imshow("Image", image); // lol how do i know if my images are greyscale

    // // allocate memory for an imaeg to be saved that is the greyscale version that it should get saved to
    // single channel 8bit color ie grey and with uchar instead of uchar4
    img_Grey.create(image.rows, image.cols, CV_8UC1);

    // struct timespec start, end;
    float milliseconds;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // // allocate matrices on host and device
    uchar4 *d_rgb, *h_rgb;
    uchar *d_grey;

    h_rgb = (uchar4*)img_RGB.ptr<uchar>(0);

    int matrixWidth = image.cols;
    int matrixHeight = image.rows;

    int numPixels = matrixWidth * matrixHeight;

    hipMalloc(&d_rgb, numPixels * sizeof(uchar4));
    hipMalloc(&d_grey, numPixels * sizeof(uchar));
    // TODO do i need memset
	// hipMemset(*d_grey, 0, sizeof(uchar) * numPixels);

    // // transfer from host to device
    hipMemcpy(d_rgb, h_rgb, numPixels * sizeof(uchar4), hipMemcpyHostToDevice);

    hipError_t code = hipGetLastError();
    if (code != hipSuccess) 
	{
        const char * errorMessage = hipGetErrorString(code);
        printf("cuda error HtoD %s", errorMessage);
    }

    // TODO check gridDim
    // call kernel to compute matrix multiplication 
    int gridDim_x = (image.cols / THREADS_DIM) + 1;
    int gridDim_y = (image.rows / THREADS_DIM) + 1;

    dim3 blockSize (THREADS_DIM, THREADS_DIM);
    dim3 gridSize (gridDim_x, gridDim_y);

    // clock_gettime(CLOCK_MONOTONIC_RAW, &start);
    hipEventRecord(start);

    greyscale <<< gridSize, blockSize >>> (d_rgb, d_grey, matrixHeight, matrixWidth, numPixels);

    // clock_gettime(CLOCK_MONOTONIC_RAW, &end);

    // // transfer back from device to host
    hipMemcpy(img_Grey.ptr<uchar>(0), d_grey, numPixels * sizeof(uchar), hipMemcpyDeviceToHost);

    code = hipGetLastError();
    if (code != hipSuccess) 
	{
        const char * errorMessage = hipGetErrorString(code);
        printf("\ncuda error DtoH %s\n\n", errorMessage);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    imwrite("grey.jpg", img_Grey);

    printf("GPU Time %f ms\n", milliseconds);

    //------------------------------------------------ DENOISE ---------------------------------------------------

    Mat denoised_image;

    denoised_image.create(image.rows, image.cols, CV_8UC1);

    uchar *d_output;

    hipMalloc(&d_output, numPixels * sizeof(uchar));
    // hipMemset(*d_output, 0, sizeof(uchar) * numPixels);
    
    denoise <<< gridSize, blockSize >>> (d_grey, d_output, matrixHeight, matrixWidth, numPixels);

    // transfer back from device to host
    hipMemcpy(denoised_image.ptr<uchar>(0), d_output, numPixels * sizeof(uchar), hipMemcpyDeviceToHost);

    imwrite("denoise.jpg", denoised_image);

    // // free memory
    hipFree(d_rgb);
    hipFree(d_grey);
    hipFree(d_output);
}